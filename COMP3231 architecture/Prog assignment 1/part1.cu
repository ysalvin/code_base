
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <fstream>
#include <chrono>

#include <stdio.h>
#include <stdlib.h>

using namespace std;

const int FILTER_WIDTH = 3;

//We will only use this filter in part 1
int FILTER[FILTER_WIDTH*FILTER_WIDTH] = {
    0, -1, 0, 
    -1, 5, -1, 
    0, -1, 0
};

// Display the first and last 10 items
// For debug only
void displayResult(const int original[], const int result[], int size) {
	cout << "Display result: ";

	for (int i = 0; i < 10; i++) {
		cout << result[i] << "\n";
	}
	cout << ".\n.\n.\n";

	for (int i = size - 10; i < size; i++) {
		cout << result[i] << "\n";
	}
}

void initData(string file, int **data, int *sizeX, int *sizeY) {
	int x;
	int y;
	long long i = 0;
	cout << "Reading "<< file << "... \n";
	ifstream myfile(file);
	if (myfile.is_open()) {
		myfile >> x;
		myfile >> y;

		int *temp = new int[x * y];
		for( i=0; i < x * y; i++){
			myfile >> temp[(int)i];
		}
		myfile.close();
		*data = temp;
		*sizeX = x;
		*sizeY = y;
	}
	else {
		cout << "ERROR: File " << file << " not found!\n";
		exit(0);
	}
	cout << i << " entries imported\n";
}

// Don't change this code
// We will evaluate your correctness based on the saved result, not printed output
void saveResult(string file, int data[], int sizeX, int sizeY) {
	long long i = 0;
	cout << "Saving data to "<< file <<"... \n";
	ofstream myfile(file, std::ofstream::out);
	if (myfile.is_open()) {
		myfile << sizeX << "\n";
		myfile << sizeY << "\n";
		for (i = 0; i < sizeX * sizeY; i++){
			myfile << data[i] << "\n";
		}
		myfile.close();
	}
	else {
		cout << "ERROR: Cannot save to " << file << "!\n";
		exit(0);
	}
	cout << i << " entries saved\n";
}

//TODO: Implement the kernel function

__global__ void sharpen(int *data, int *result, int *sizeX, int *sizeY, int *FILTER){
	// use block index to locate the image pixel
	int image_pixel = threadIdx.x + blockIdx.x * blockDim.x;;
	// use thread index to locate the filer pixel
	// int flter_pixel = blockIdx.x;
	int x = image_pixel % (*sizeX);
    int y = image_pixel / (*sizeX);

	if (x < (*sizeX) && y < (*sizeY)){
		int sum = 0;

		for (int i = 0; i < FILTER_WIDTH; ++i) {
			for (int j = 0; j < FILTER_WIDTH; ++j) {
				// for each loop of filter, find the center of the filter
				int centerX = x - FILTER_WIDTH / 2 + j;
				int centerY = y - FILTER_WIDTH / 2 + i;

				// add to the sum only if the filter center is within the image boundary, which is possible in reality
				if (centerX >= 0 && centerX < (*sizeX) && centerY >= 0 && centerY < (*sizeY)) {
					sum += data[centerY * (*sizeX) + centerX ] * FILTER[i * FILTER_WIDTH + j ];
				}
			}
		}
		result [y * (*sizeX) + x] = sum;
	}
}

// GPU implementation
void GPU_Test(int data[], int result[], int sizeX, int sizeY) {
	// input:
	//	int data[] - int array holding the flattened original image
	//	int sizeX - the width of the image
	//	int sizeY - the height of the image
	// output:
	//	int result[] - int array holding the output image

	// each threads for a signal filter 
	const int BLOCK_SIZE =256;
	// each pixel on the image is a block
	const int n_blocks = ((sizeX*sizeY)/BLOCK_SIZE);

	// TODO: malloc memory, copy input

	// declare device copies 
	int *d_data;
	int *d_result;
	int *d_sizeX, *d_sizeY;
	int *d_FILTER;

	// allocate space fro device copies of 
	hipMalloc((void **) &d_result, sizeX * sizeY * sizeof(int));
	hipMalloc((void **) &d_data, sizeX * sizeY * sizeof(int));
	hipMalloc((void **) &d_sizeX, sizeof(int));
    hipMalloc((void **) &d_sizeY, sizeof(int));
	hipMalloc((void **) &d_FILTER, FILTER_WIDTH * FILTER_WIDTH * sizeof(int));

	// Copy data to device
	hipMemcpy(d_data, data, sizeof(int) * sizeX * sizeY, hipMemcpyHostToDevice);
	hipMemcpy(d_result, result, sizeof(int) * sizeX * sizeY, hipMemcpyHostToDevice);
	hipMemcpy(d_sizeX, &sizeX, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sizeY, &sizeY, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_FILTER, FILTER, FILTER_WIDTH * FILTER_WIDTH * sizeof(int), hipMemcpyHostToDevice);

	// Start timer for kernel
	// Don't change this function
	auto startKernel = chrono::steady_clock::now();

	// TODO: call the kernel function
	sharpen<<<n_blocks, BLOCK_SIZE>>>(d_data, d_result, d_sizeX, d_sizeY, d_FILTER);
	// End timer for kernel and display kernel time
	hipDeviceSynchronize(); // <- DO NOT REMOVE
	auto endKernel = chrono::steady_clock::now();
	cout << "Kernel Elapsed time: " << chrono::duration <double, milli>(endKernel - startKernel).count() << "ms\n";

	// TODO: copy result from device to host
	hipMemcpy(result, d_result, sizeof(int) * sizeX * sizeY, hipMemcpyDeviceToHost);

	// TODO: free device memory <- important, keep your code clean
	hipFree(d_data);
	hipFree(d_result);
	hipFree(d_sizeX);
	hipFree(d_sizeY);
	hipFree(d_FILTER);
}


// CPU implementation
void CPU_Test(int data[], int result[], int sizeX, int sizeY) {
	// input:
	//	int data[] - int array holding the flattened original image
	//	int sizeX - the width of the image
	//	int sizeY - the height of the image
	// output:
	//	int result[] - int array holding the output image

	// TODO: sharpen the image with filter
	//       apply zero padding for the border

	// loop the y axis first, then the x axis
	for (int y=0; y<sizeY; y++){
		for (int x=0; x<sizeX; x++){
			int sum = 0;

			// for every cell, iterate over the filter
			 for (int i = 0; i < FILTER_WIDTH; ++i) {
                for (int j = 0; j < FILTER_WIDTH; ++j) {
                    // for each loop of filter, find the center of the filter
                    int centerX = x - FILTER_WIDTH / 2 + j;
                    int centerY = y - FILTER_WIDTH / 2 + i;

                    // add to the sum only if the filter center is within the image boundary, which is possible in reality
                    if (centerX >= 0 && centerX < sizeX && centerY >= 0 && centerY < sizeY) {
                        sum += data[centerY * sizeX + centerX ] * FILTER[i * FILTER_WIDTH + j ];
                    }
                }
            }
			result [y * sizeX + x] = sum;
		}
	}
}

// The input is a 2D grayscale image
// The image is flattened into a text file of pixel values.
int main(int argc, char *argv[]) {
	string inputFile = (argc == 1) ? "image_grey.txt" : argv[1];

	int sizeX;
	int sizeY;
	int *dataForCPUTest;
	int *dataForGPUTest;	

	initData(inputFile, &dataForCPUTest, &sizeX, &sizeY);
	initData(inputFile, &dataForGPUTest, &sizeX, &sizeY);

	int size = sizeX * sizeY;
	int *resultForCPUTest = new int[size];
	int *resultForGPUTest = new int[size];

	cout << "\n";

	cout << "CPU Implementation\n";

	auto startCPU = chrono::steady_clock::now();
	CPU_Test(dataForCPUTest, resultForCPUTest, sizeX, sizeY);
	auto endCPU = chrono::steady_clock::now();

	cout << "Elapsed time: " << chrono::duration <double, milli>(endCPU - startCPU).count() << "ms\n";
	// For debug
	// displayResult(dataForCPUTest, resultForCPUTest, size);

	saveResult("grey_result_CPU.txt",resultForCPUTest, sizeX, sizeY);

	cout << "\n";
	cout << "GPU Implementation\n";

	auto startGPU = chrono::steady_clock::now();
	GPU_Test(dataForGPUTest, resultForGPUTest, sizeX, sizeY);
	auto endGPU = chrono::steady_clock::now();

	cout << "Elapsed time: " << chrono::duration <double, milli>(endGPU - startGPU).count() << "ms\n";

	// For debug
	// displayResult(dataForGPUTest, resultForGPUTest, size);
	
	saveResult("grey_result_GPU.txt",resultForGPUTest, sizeX, sizeY);

	return 0;
}


