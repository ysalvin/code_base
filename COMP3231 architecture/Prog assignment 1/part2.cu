
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <fstream>
#include <chrono>

#include <stdio.h>
#include <stdlib.h>

using namespace std;

//We may change this value!!
const int FILTER_WIDTH = 7;
const int BLOCK_SIZE = 512 ;


// We may change this value!!!
int FILTER[FILTER_WIDTH*FILTER_WIDTH] = {
	1,4,7,10,7,4,1,
	4,12,26,33,26,12,4,
	7,26,55,71,55,26,7,
	10,33,71,91,71,33,10,
	7,26,55,71,55,26,7,
	4,12,26,33,26,12,4,
	1,4,7,10,7,4,1
};

// Display the first and last 10 items
// For debug only
void displayResult(const int original[], const int result[], int size) {
	cout << "Display result: ";
	cout << "(original -> result)\n";

	for (int i = 0; i < 10; i++) {
		cout << original[i] << " -> " << result[i] << "\n";
	}
	cout << ".\n.\n.\n";

	for (int i = size - 10; i < size; i++) {
		cout << original[i] << " -> " << result[i] << "\n";
	}
}

void initColorData(string file, int **data, int *sizeX, int *sizeY) {
	int x;
	int y;
	long long i = 0;
	cout << "Reading "<< file << "... \n";
	ifstream myfile(file);
	if (myfile.is_open()) {
		myfile >> x;
		myfile >> y;

		int *temp = new int[x * y * 3];
		for( i=0; i < x * y * 3; i++){
			myfile >> temp[(int)i];
		}
		myfile.close();
		*data = temp;
		*sizeX = x;
		*sizeY = y;
	}
	else {
		cout << "ERROR: File " << file << " not found!\n";
		exit(0);
	}
	cout << i << " entries imported\n";
}

void saveResult(string file, int data[], int sizeX, int sizeY) {
	long long i = 0;
	cout << "Saving data to "<< file <<"... \n";
	ofstream myfile(file, std::ofstream::out);
	if (myfile.is_open()) {
		myfile << sizeX << "\n";
		myfile << sizeY << "\n";
		for (i = 0; i < sizeX * sizeY; i++){
			myfile << data[3* i] << " " << data[3* i + 1] << " " << data[3* i+ 2]<< "\n";
		}
		myfile.close();
	}
	else {
		cout << "ERROR: Cannot save to " << file << "!\n";
		exit(0);
	}
	cout << i << " entries saved\n";
}

// TODO: implement the kneral function for 2D smoothing 



__global__ void sharpen(int *data, int *result, int sizeX, int sizeY, int *FILTER){
    // use block index to locate the image pixel
    int image_pixel = threadIdx.x + blockIdx.x * blockDim.x;
    int x = image_pixel % sizeX;
    int y = image_pixel / sizeX;

    if (x < sizeX && y < sizeY){
        int sumR = 0;
        int sumG = 0;
        int sumB = 0;

        for (int i = 0; i < FILTER_WIDTH; ++i) {
            for (int j = 0; j < FILTER_WIDTH; ++j) {
                // for each loop of filter, find the center of the filter
                int centerX = x - FILTER_WIDTH / 2 ;
                int centerY = y - FILTER_WIDTH / 2 ;

				if (centerX >= 0 && centerX < sizeX - 1) {
					centerX += j;
				}
				if (centerY >= 0 && centerY < sizeY - 1) {
					centerY += i;
				}
				int filter_value =  FILTER[i * FILTER_WIDTH + j];
                // add to the sum only if the filter center is within the image boundary
                if (centerX >= 0 && centerX < sizeX && centerY >= 0 && centerY < sizeY) {
                    sumR += data[(centerY * sizeX * 3) + centerX * 3] *	filter_value;
                    sumG += data[(centerY * sizeX * 3) + centerX * 3 + 1] * filter_value;
                    sumB += data[(centerY * sizeX * 3) + centerX * 3 + 2] * filter_value;
                }
            }
        }

        result [y * sizeX * 3 + x * 3 ] = sumR;
        result [y * sizeX * 3 + x * 3 + 1] = sumG;
        result [y * sizeX * 3 + x * 3 + 2] = sumB;
    }
}

void normalize_output(int result[], int sizeX, int sizeY){
	int size = sizeX * sizeY;
	int *scaled =  (int * ) malloc(sizeof(int)*size*3);
	for (int i=0; i<3; i++){
		int source_min = INT_MAX;
		int source_max = 0;
		for (int j=0; j<size; j++){
			if (result[j*3+i] < source_min)
            	source_min = result[j*3+i];
			if (result[j*3+i] > source_max)
				source_max = result[j*3+i];
		}
		int source_scale = source_max - source_min;
		for (int j=0; j<size; j++) {
			int zsrc = result[j*3+i] - source_min;

			scaled[j*3+i] = zsrc * 255 / source_scale;
    	}
	}
	memcpy(result, scaled, sizeof(int)*size*3);
}


// GPU implementation
void GPU_Test(int data[], int result[], int sizeX, int sizeY) {
	// input:
	//	int data[] - int array holding the flattened original image
	//	int sizeX - the width of the image
	//	int sizeY - the height of the image
	// output:
	//	int result[] - int array holding the image

	// each threads for a signal filter 
	// const int BLOCK_SIZE = 512 ;
	// each pixel on the image is a block
	const int n_blocks = ((sizeX*sizeY)/BLOCK_SIZE);
	const int data_size = sizeX * sizeY * sizeof(int) * 3;

	// TODO: allocate device memory and copy data onto the device

	// declare device copies 
	int *d_data;
	int *d_result;
	int *d_FILTER;

	// allocate space fro device copies of 
	hipMalloc((void **) &d_result, data_size);
	hipMalloc((void **) &d_data, data_size);
	hipMalloc((void **) &d_FILTER, FILTER_WIDTH * FILTER_WIDTH * sizeof(int));

	// Copy data to device
	hipMemcpy(d_data, data, data_size, hipMemcpyHostToDevice);
	hipMemcpy(d_result, result, data_size, hipMemcpyHostToDevice);
	hipMemcpy(d_FILTER, FILTER, FILTER_WIDTH * FILTER_WIDTH * sizeof(int), hipMemcpyHostToDevice);

	// Start timer for kernel
	auto startKernel = chrono::steady_clock::now();

	// TODO: call the kernel function
	sharpen<<<n_blocks, BLOCK_SIZE>>>(d_data, d_result, sizeX, sizeY, d_FILTER);
	// End timer for kernel and display kernel time
	hipDeviceSynchronize(); // <- DO NOT REMOVE
	auto endKernel = chrono::steady_clock::now();
	cout << "Kernel Elapsed time: " << chrono::duration <double, milli>(endKernel - startKernel).count() << "ms\n";

	// TODO: copy result from device to host
	hipMemcpy(result, d_result, data_size, hipMemcpyDeviceToHost);
	normalize_output(result, sizeX, sizeY);

	// TODO: free device memory <- important, keep your code clean
	hipFree(d_data);
	hipFree(d_result);
	hipFree(d_FILTER);

}


// CPU implementation
void CPU_Test(int data[], int result[], int sizeX, int sizeY) {
	// input:
	//	int data[] - int array holding the flattened original image
	//	int sizeX - the width of the image
	//	int sizeY - the height of the image
	// output:
	//	int result[] - int array holding the image

	// TODO: smooth the image with filter size = FILTER_WIDTH
	//       apply zero padding for the border

	for (int y=0; y<sizeY; y++){
		for (int x=0; x<sizeX; x++){
			int sumR = 0;
			int sumG = 0;
			int sumB = 0;

			// for every cell, iterate over the filter
			 for (int i = 0; i < FILTER_WIDTH; ++i) {
                for (int j = 0; j < FILTER_WIDTH; ++j) {
                    // for each loop of filter, find the center of the filter
                    int centerX = x - FILTER_WIDTH / 2;
                    int centerY = y - FILTER_WIDTH / 2;

					if (centerX >= 0 && centerX < sizeX - 1) {
						centerX += j;
					}
					if (centerY >= 0 && centerY < sizeY - 1) {
						centerY += i;
					}

                    // add to the sum only if the filter center is within the image boundary, which is possible in reality
                    if (centerX >= 0 && centerX < sizeX && centerY >= 0 && centerY < sizeY) {
                        // sum += data[centerY * sizeX + centerX ] * FILTER[i * FILTER_WIDTH + j ];
						sumR += data[(centerY * sizeX *3 ) + centerX * 3 ] * FILTER[i * FILTER_WIDTH + j ];
						sumG += data[(centerY * sizeX *3 ) + centerX * 3 + 1] * FILTER[i * FILTER_WIDTH + j ];
						sumB += data[(centerY * sizeX *3 ) + centerX * 3 + 2] * FILTER[i * FILTER_WIDTH + j ];

                    }
                }
            }
			result [y * sizeX * 3 + x * 3 ] = sumR;
        	result [y * sizeX * 3 + x * 3 + 1] = sumG;
        	result [y * sizeX * 3 + x * 3 + 2] = sumB;
		}
	}
	normalize_output(result, sizeX, sizeY);
}

// The image is flattened into a text file of pixel values.
int main(int argc, char *argv[]) {
	string inputFile = (argc == 1) ? "image_color.txt" : argv[1];

	int sizeX;
	int sizeY;
	int *dataForCPUTest;
	int *dataForGPUTest;	

	initColorData(inputFile, &dataForCPUTest, &sizeX, &sizeY);
	initColorData(inputFile, &dataForGPUTest, &sizeX, &sizeY);

	int size = sizeX * sizeY * 3;
	int *resultForCPUTest = new int[size];
	int *resultForGPUTest = new int[size];

	cout << "\n";

	cout << "CPU Implementation\n";

	auto startCPU = chrono::steady_clock::now();
	CPU_Test(dataForCPUTest, resultForCPUTest, sizeX, sizeY);
	auto endCPU = chrono::steady_clock::now();

	cout << "Elapsed time: " << chrono::duration <double, milli>(endCPU - startCPU).count() << "ms\n";

	// displayResult(dataForCPUTest, resultForCPUTest, size);

	saveResult("color_result_CPU.txt",resultForCPUTest, sizeX, sizeY);

	cout << "\n";
	cout << "GPU Implementation\n";

	auto startGPU = chrono::steady_clock::now();
	GPU_Test(dataForGPUTest, resultForGPUTest, sizeX, sizeY);
	auto endGPU = chrono::steady_clock::now();

	cout << "Elapsed time: " << chrono::duration <double, milli>(endGPU - startGPU).count() << "ms\n";

	// displayResult(dataForGPUTest, resultForGPUTest, size);
	saveResult("color_result_GPU.txt",resultForGPUTest, sizeX, sizeY);

	return 0;
}
